#include "hip/hip_runtime.h"
//
//  vec.cpp
//  CUDA Physics
//
//  Created by Jacob Austin on 5/13/18.
//  Copyright © 2018 Jacob Austin. All rights reserved.
//

#include "vec.h"

#if __CUDA_ARCH__ < 600
__device__ double atomicDoubleAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
            (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__ double atomicDoubleExch(double* address, double val)
{
    unsigned long long int* address_as_ull =
            (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

CUDA_DEVICE void Vec::atomicVecAdd(const Vec & v) {
atomicDoubleAdd(&data[0], (double) v.data[0]);
atomicDoubleAdd(&data[1], (double) v.data[1]);
atomicDoubleAdd(&data[2], (double) v.data[2]);
}

CUDA_DEVICE void Vec::atomicVecExch(const Vec &v) {
    atomicDoubleExch(&data[0], (double) v.data[0]);
    atomicDoubleExch(&data[1], (double) v.data[1]);
    atomicDoubleExch(&data[2], (double) v.data[2]);
}

Vec::Vec() {
        data[0] = 0;
        data[1] = 0;
    	data[2] = 0;
} // default

Vec::Vec(const Vec & v) {
    data[0] = v.data[0];
    data[1] = v.data[1];
    data[2] = v.data[2];
} // copy constructor

Vec::Vec(double x, double y, double z) {
    data[0] = x;
    data[1] = y;
    data[2] = z;
} // initialization from x, y, and z values

Vec & Vec::operator=(const Vec & v) {
    if (this == &v) {
        return *this;
    }

    data[0] = v.data[0];
    data[1] = v.data[1];
    data[2] = v.data[2];

    return *this;
}

Vec & Vec::operator+=(const Vec & v) {
    data[0] += v.data[0];
    data[1] += v.data[1];
    data[2] += v.data[2];
    return *this;
}

Vec & Vec::operator-=(const Vec & v) {
    data[0] -= v.data[0];
    data[1] -= v.data[1];
    data[2] -= v.data[2];
    return *this;
}

Vec Vec::operator-() const{
    return Vec(-data[0], -data[1], -data[2]);
}

double & Vec::operator [] (int n) {
    if (n < 0 || n >= 3) {
        printf("%s\n", "Out of bounds!");
        return data[0];
    } else {
        return data[n];
    }
}

const double & Vec::operator [] (int n) const {
    if (n < 0 || n >= 3) {
        printf("%s\n", "Out of bounds!");
        return data[0];
    } else {
        return data[n];
    }
}

Vec operator+(const Vec & v1, const Vec & v2) {
    return Vec(v1.data[0] + v2.data[0], v1.data[1] + v2.data[1], v1.data[2] + v2.data[2]);
}

Vec operator-(const Vec & v1, const Vec & v2) {
    return Vec(v1.data[0] - v2.data[0], v1.data[1] - v2.data[1], v1.data[2] - v2.data[2]);
}

Vec operator*(const double x, const Vec & v) {
    return Vec(v.data[0] * x, v.data[1] * x, v.data[2] * x);
}

Vec operator*(const Vec & v, const double x) {
    return x * v;
}

bool operator==(const Vec & v1, const Vec & v2) {
    return (v1[0] == v2[0] && v1[1] == v2[1] && v1[2] == v2[2]);
}

bool operator<(const Vec &v1, const Vec &v2) {
    return ((v1[0] < v2[0]) || (v1[0] <= v2[0] && v1[1] < v2[1])
            || (v1[0] <= v2[0] && v1[1] <= v2[1] && v1[2] < v2[2]));
}

bool operator>(const Vec &v1, const Vec &v2) {
    return !(v1<v2);
}

Vec operator*(const Vec & v1, const Vec & v2) {
    return Vec(v1.data[0] * v2.data[0], v1.data[1] * v2.data[1], v1.data[2] * v2.data[2]);
} // Multiplies two Vecs (elementwise)

Vec operator/(const Vec & v, const double x) {
    return Vec(v.data[0] / x, v.data[1] / x, v.data[2] / x);
} //  vector over double

Vec operator/(const Vec & v1, const Vec & v2) {
    return Vec(v1.data[0] / v2.data[0], v1.data[1] / v2.data[1], v1.data[2] / v2.data[2]);
} // divides two Vecs (elementwise)

std::ostream & operator << (std::ostream & strm, const Vec & v) {
    return strm << "(" << v[0] << ", " << v[1] << ", " << v[2] << ")";
} // print

void Vec::print() {
    printf("(%3f, %3f, %3f)\n", data[0], data[1], data[2]);
}

double Vec::norm() const {
    return sqrt(pow(data[0], 2) + pow(data[1], 2) + pow(data[2], 2));
} // gives vector norm

double Vec::sum() const {
    return data[0] + data[1] + data[2];
} // sums all components of the vector

Vec Vec::normalized() const {
    double l = norm();
    return l > 0 ? (*this)/l : (*this);
}

CUDA_CALLABLE_MEMBER double dot(const Vec & a, const Vec & b) {
    return (a * b).sum();
}





Vec & Vec::operator+=(double x) {
    data[0] = data[0]+=x;
    data[1] = data[1]+=x;
    data[2] = data[2]+=x;
    return *this;
}

Vec & Vec::operator-=(double x) {
    data[0] = data[0]-=x;
    data[1] = data[1]-=x;
    data[2] = data[2]-=x;
    return *this;
}

Vec & Vec::operator*=(double x) {
    data[0] = data[0]*=x;
    data[1] = data[0]*=x;
    data[2] = data[0]*=x;
    return *this;
}

Vec & Vec::operator/=(double x) {
    data[0] = data[0]/=x;
    data[1] = data[0]/=x;
    data[2] = data[0]/=x;
    return *this;
}
