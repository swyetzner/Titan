#include "hip/hip_runtime.h"
//
//  complexVec.cpp
//  CUDA Physics
//
//  Created by Salvy Cavicchio on 10/11/20.
//  Copyright © 2020 Salvy Cavicchio. All rights reserved.
//

#include "complexVec.h"

// COMPLEX VECTOR

ComplexVec::ComplexVec() {
    data[0] = make_hipDoubleComplex(0, 0);
    data[1] = make_hipDoubleComplex(0, 0);
   	data[2] = make_hipDoubleComplex(0, 0);
} // default

ComplexVec::ComplexVec(const ComplexVec & v) {
    data[0] = v.data[0];
    data[1] = v.data[1];
    data[2] = v.data[2];
} // copy constructor

ComplexVec::ComplexVec(hipDoubleComplex x, hipDoubleComplex y, hipDoubleComplex z) {
    data[0] = x;
    data[1] = y;
    data[2] = z;
} // initialization from x, y, and z values

ComplexVec & ComplexVec::operator=(const ComplexVec & v) {
    if (this == &v) {
        return *this;
    }

    data[0] = v.data[0];
    data[1] = v.data[1];
    data[2] = v.data[2];

    return *this;
}

ComplexVec & ComplexVec::operator+=(const ComplexVec & v) {
	data[0] = hipCadd(data[0],v.data[0]);
    data[1] = hipCadd(data[1],v.data[1]);
    data[2] = hipCadd(data[2],v.data[2]);
    return *this;
}

ComplexVec & ComplexVec::operator+=(const Vec & v) {
    data[0] = hipCadd(data[0],make_hipDoubleComplex(v[0], 0));
    data[1] = hipCadd(data[1],make_hipDoubleComplex(v[1], 0));
    data[2] = hipCadd(data[2],make_hipDoubleComplex(v[2], 0));
    return *this;
}

ComplexVec & ComplexVec::operator-=(const ComplexVec & v) {
	data[0] = hipCsub(data[0],v.data[0]);
    data[1] = hipCsub(data[1],v.data[1]);
    data[2] = hipCsub(data[2],v.data[2]);
    return *this;
}

ComplexVec & ComplexVec::operator-=(const Vec & v) {
    data[0] = hipCsub(data[0], make_hipDoubleComplex(v.data[0], 0));
    data[1] = hipCsub(data[1], make_hipDoubleComplex(v.data[1], 0));
    data[2] = hipCsub(data[2], make_hipDoubleComplex(v.data[2], 0));
    return *this;
}

ComplexVec & ComplexVec::operator*=(const ComplexVec & v) {
	data[0] = hipCmul(data[0],v.data[0]);
    data[1] = hipCmul(data[1],v.data[1]);
    data[2] = hipCmul(data[2],v.data[2]);
    return *this;
}

ComplexVec & ComplexVec::operator*=(const Vec & v) {
    data[0] = hipCmul(data[0],make_hipDoubleComplex(v[0], 0));
    data[1] = hipCmul(data[1],make_hipDoubleComplex(v[1], 0));
    data[2] = hipCmul(data[2],make_hipDoubleComplex(v[2], 0));
    return *this;
}

ComplexVec & ComplexVec::operator/=(const ComplexVec & v) {
	data[0] = hipCdiv(data[0],v.data[0]);
    data[1] = hipCdiv(data[1],v.data[1]);
    data[2] = hipCdiv(data[2],v.data[2]);
    return *this;
}

ComplexVec & ComplexVec::operator+=(double x) {
	data[0] = hipCadd(data[0],make_hipDoubleComplex(x, 0));
	data[1] = hipCadd(data[1],make_hipDoubleComplex(x, 0));
	data[2] = hipCadd(data[2],make_hipDoubleComplex(x, 0));
	return *this;
}

ComplexVec & ComplexVec::operator-=(double x) {
	data[0] = hipCsub(data[0],make_hipDoubleComplex(x, 0));
	data[1] = hipCsub(data[1],make_hipDoubleComplex(x, 0));
	data[2] = hipCsub(data[2],make_hipDoubleComplex(x, 0));
	return *this;
}

ComplexVec & ComplexVec::operator*=(double x) {
	data[0] = hipCmul(data[0],make_hipDoubleComplex(x, 0));
	data[1] = hipCmul(data[1],make_hipDoubleComplex(x, 0));
	data[2] = hipCmul(data[2],make_hipDoubleComplex(x, 0));
	return *this;
}

ComplexVec & ComplexVec::operator/=(double x) {
	data[0] = hipCdiv(data[0],make_hipDoubleComplex(x, 0));
	data[1] = hipCdiv(data[1],make_hipDoubleComplex(x, 0));
	data[2] = hipCdiv(data[2],make_hipDoubleComplex(x, 0));
	return *this;
}

//CUDA_CALLABLE_MEMBER ComplexVec operator-() const; // returns the negative -z = -a -bi
hipDoubleComplex & ComplexVec::operator [] (int n) {
	if (n < 0 || n >= 3) {
        printf("%s\n", "Out of bounds!");
        return data[0];
    } else {
        return data[n];
    }
}

const hipDoubleComplex & ComplexVec::operator [] (int n) const {
	if (n < 0 || n >= 3) {
        printf("%s\n", "Out of bounds!");
        return data[0];
    } else {
        return data[n];
    }
}

bool operator==(const ComplexVec & v1, const ComplexVec & v2) {
	return ((v1[0].x == v2[0].x && v1[1].x == v2[1].x && v1[2].x == v2[2].x) && (v1[0].y == v2[0].y && v1[1].y == v2[1].y && v1[2].y == v2[2].y));
}

ComplexVec operator+(const ComplexVec & v1, const ComplexVec & v2) {
	return ComplexVec(v1+v2);
}

ComplexVec operator+(const ComplexVec & v1, const Vec & v2) {
    return ComplexVec(v1+v2);
}

ComplexVec operator-(const ComplexVec & v1, const ComplexVec & v2) {
	return ComplexVec(v1-v2);
}

ComplexVec operator*(const ComplexVec & v1, const ComplexVec & v2) {
	return ComplexVec(v1*v2);
}

ComplexVec operator*(const ComplexVec & v1, const Vec & v2) {
    return ComplexVec(v1*v2);
}

ComplexVec operator/(const ComplexVec & v1, const ComplexVec & v2) {
	return ComplexVec(v1/v2);
}

std::ostream & operator << (std::ostream & strm, const ComplexVec & v) {
	return strm << "(" << v[0].x << " + " << v[0].y << "i" << ", " << v[1].x << " + " << v[0].y << "i" << ", " << v[2].x << " + " << v[0].y << "i)";
}

void ComplexVec::print() {
	printf("(%3f + %3fi, %3f + %3fi, %3f + %3fi)\n", data[0].x, data[0].y, data[1].x, data[1].y, data[2].x, data[2].y);
}

hipDoubleComplex ComplexVec::sum() const {
	return hipCadd(hipCadd(data[0],data[1]),data[2]);
}

ComplexVec ComplexVec::exp() {
    hipDoubleComplex dataNew[3] = { 0 };
    for (int i=0; i<3; i++) {
        double s = 0, c = 0, m = 0;
        sincos(data[i].y, &s, &c);
        m = std::exp(data[i].x);
        dataNew[i] = make_hipDoubleComplex(m*c, m*s);
    }
    return ComplexVec(dataNew[0], dataNew[1], dataNew[2]);
}

Vec ComplexVec::realSign() {
    double dataNew[3] = { 0 };
    for (int i=0; i<3; i++) {
        dataNew[i] = data[i].x < 0 ? -1 : 1;
    }
    return Vec(dataNew[0], dataNew[1], dataNew[2]);
}

Vec ComplexVec::abs() {
    double dataNew[3] = { 0 };
    for (int i = 0; i<3; i++) {
        dataNew[i] = sqrt(data[i].x*data[i].x + data[i].y*data[i].y);
    }
    return Vec(dataNew[0], dataNew[1], dataNew[2]);
}

CUDA_CALLABLE_MEMBER hipDoubleComplex dot(const ComplexVec & a, const ComplexVec & b) {
    return (a * b).sum();
}

